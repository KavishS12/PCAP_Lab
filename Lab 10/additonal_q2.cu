#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16  
#define MAX_MASK_WIDTH 10  

__constant__ int d_mask[MAX_MASK_WIDTH];  

__global__ void tiled_convolution_1D(int *d_input, int *d_output, int width, int mask_width) {
    extern __shared__ int shared_mem[];  
    int radius = mask_width / 2;
    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tx;

    int shared_idx = tx + radius;
    
    if (idx < width) {
        shared_mem[shared_idx] = d_input[idx];
    } else {
        shared_mem[shared_idx] = 0;
    }

    if (tx < radius) {
        shared_mem[tx] = (idx - radius >= 0) ? d_input[idx - radius] : 0;
        shared_mem[shared_idx + TILE_SIZE] = (idx + TILE_SIZE < width) ? d_input[idx + TILE_SIZE] : 0;
    }

    __syncthreads();

    if (idx < width) {
        int sum = 0;
        for (int i = -radius; i <= radius; i++) {
            sum += shared_mem[shared_idx + i] * d_mask[i + radius];
        }
        d_output[idx] = sum;
    }
}

int main() {
    int width, mask_width;

    printf("Enter the size of the input array: ");
    scanf("%d", &width);
    printf("Enter the mask width (odd number, max %d): ", MAX_MASK_WIDTH);
    scanf("%d", &mask_width);
    
    if (mask_width > MAX_MASK_WIDTH || mask_width % 2 == 0) {
        printf("Invalid mask size! It must be an odd number <= %d.\n", MAX_MASK_WIDTH);
        return 1;
    }

    int input[width], output[width], mask[mask_width];
    int *d_input, *d_output;

    printf("Enter the input array:\n");
    for (int i = 0; i < width; i++) {
        scanf("%d", &input[i]);
    }

    printf("Enter mask:\n");
    for (int i = 0; i < mask_width; i++) {
        scanf("%d", &mask[i]);
    }

    hipMalloc((void**)&d_input, width * sizeof(int));
    hipMalloc((void**)&d_output, width * sizeof(int));

    hipMemcpy(d_input, input, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask, mask_width * sizeof(int));

    int blocks = (width + TILE_SIZE - 1) / TILE_SIZE;
    int shared_mem_size = (TILE_SIZE + 2 * (mask_width / 2)) * sizeof(int);
    
    tiled_convolution_1D<<<blocks, TILE_SIZE, shared_mem_size>>>(d_input, d_output, width, mask_width);

    hipMemcpy(output, d_output, width * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nConvolved Output:\n");
    for (int i = 0; i < width; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}