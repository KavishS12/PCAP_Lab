#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 8   
#define K 3   
#define THREADS_PER_BLOCK 16

__constant__ int d_kernel[K];

__global__ void convolution_1D(int *d_arr, int *d_res, int width) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= width) return;

    int sum = 0.0;
    int kernel_radius = K / 2;
    for (int i = -kernel_radius; i <= kernel_radius; i++) {
        int index = idx + i;
        if (index >= 0 && index < width) {
            sum += d_arr[index] * d_kernel[i + kernel_radius];
        }
    }
    
    d_res[idx] = sum;
}

int main() {
    int arr[N], res[N], kernel[K];
    int *d_arr, *d_res;

    printf("Enter the array : ");
    for(int i=0;i<N;i++){
        scanf("%d",&arr[i]);
    }

    printf("Enter the kernel : ");
    for(int i=0;i<K;i++){
        scanf("%d",&kernel[i]);
    }

    hipMalloc((void**)&d_arr, N * sizeof(int));
    hipMalloc((void**)&d_res, N * sizeof(int));

    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), kernel, K * sizeof(int));

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    convolution_1D<<<blocks, THREADS_PER_BLOCK>>>(d_arr, d_res, N);

    hipMemcpy(res, d_res, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Input Array:\n");
    for (int i = 0; i < N; i++) printf("%d ",arr[i]);
    printf("\n\nConvolved Output:\n");
    for (int i = 0; i < N; i++) printf("%d ",res[i]);
    printf("\n");

    hipFree(d_arr);
    hipFree(d_res);

    return 0;
}
