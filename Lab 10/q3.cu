#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void inclusive_scan(int *arr, int *res, int n) {
    extern __shared__ int temp[];
    int thid = threadIdx.x;

    int index1 = 2 * thid;
    int index2 = 2 * thid + 1;

    if (index1 < n) temp[index1] = arr[index1];
    else temp[index1] = 0;

    if (index2 < n) temp[index2] = arr[index2];
    else temp[index2] = 0;

    __syncthreads();

    // Up-sweep (reduce)
    for (int d = 1; d < n; d *= 2) {
        __syncthreads();
        int k = (thid + 1) * d * 2 - 1;
        if (k < n) {
            temp[k] += temp[k - d];
        }
    }

    __syncthreads();

    // Down-sweep (inclusive scan)
    for (int d = n / 2; d > 0; d /= 2) {
        __syncthreads();
        int k = (thid + 1) * d * 2 - 1;
        if (k + d < n) {
            temp[k + d] += temp[k];
        }
    }

    __syncthreads();
    if (index1 < n) res[index1] = temp[index1];
    if (index2 < n) res[index2] = temp[index2];
}

int main() {
    int N;
    printf("Enter N: ");
    scanf("%d", &N);

    int *arr = (int *)malloc(N * sizeof(int));
    int *res = (int *)malloc(N * sizeof(int));
    int *d_arr, *d_res;

    printf("Enter the array: ");
    for (int i = 0; i < N; i++) {
        scanf("%d", &arr[i]);
    }

    hipMalloc((void**)&d_arr, N * sizeof(int));
    hipMalloc((void**)&d_res, N * sizeof(int));
    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = (N + 1) / 2;
    inclusive_scan<<<1, threadsPerBlock, 2 * N * sizeof(int)>>>(d_arr, d_res, N);

    hipMemcpy(res, d_res, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Resultant Array after Inclusive Scan: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", res[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_res);
    free(arr);
    free(res);

    return 0;
}
