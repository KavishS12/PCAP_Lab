#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add_vectors1(int *v1,int *v2,int *res,int N){
    int i = blockIdx.x;
    if (i<N) {
        res[i] = v1[i] + v2[i];
    }
}

__global__ void add_vectors2(int *v1,int *v2,int *res,int N){
    int i = threadIdx.x;
    if (i<N) {
        res[i] = v1[i] + v2[i];
    }
}

int main(void)
{
    int N;
    printf("Enter the value of N : ");
    scanf("%d",&N);

    int *v1 = (int*)malloc(N*sizeof(int));    
    int *v2 = (int*)malloc(N*sizeof(int));
    int *res = (int*)malloc(N*sizeof(int));

    int *d_v1,*d_v2,*d_res;
    hipMalloc((void**)&d_v1,N*sizeof(int));
    hipMalloc((void**)&d_v2,N*sizeof(int));
    hipMalloc((void**)&d_res,N*sizeof(int));

    for(int i=0;i<N;i++){
        v1[i] = i;
        v2[i] = 2*i;
    }

    hipMemcpy(d_v1,v1,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_v2,v2,N*sizeof(int),hipMemcpyHostToDevice);  

    add_vectors1<<<N,1>>>(d_v1,d_v2,d_res,N);
    hipMemcpy(res,d_res,N*sizeof(int),hipMemcpyDeviceToHost);
    printf("Resultant vector using N blocks : ");
    for(int i=0;i<N;i++){
        printf("%d ",res[i]);
    }
    printf("\n");

    add_vectors2<<<N,1>>>(d_v1,d_v2,d_res,N);
    hipMemcpy(res,d_res,N*sizeof(int),hipMemcpyDeviceToHost);
    printf("Resultant vector using N threads : ");
    for(int i=0;i<N;i++){
        printf("%d ",res[i]);
    }
    printf("\n");

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_res);

    free(v1);
    free(v2);
    free(res);

    return 0;
}
