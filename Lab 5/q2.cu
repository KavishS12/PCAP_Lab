#include "hip/hip_runtime.h"

#include<stdio.h>

#define THREADS_PER_BLOCK 256

__global__ void add_vectors(int *v1,int *v2,int *res,int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<N) {
        res[i] = v1[i] + v2[i];
    }
}

int main(void)
{
    int N;
    printf("Enter the value of N : ");
    scanf("%d",&N);

    int *v1 = (int*)malloc(N*sizeof(int));    
    int *v2 = (int*)malloc(N*sizeof(int));
    int *res = (int*)malloc(N*sizeof(int));

    int *d_v1,*d_v2,*d_res;
    hipMalloc((void**)&d_v1,N*sizeof(int));
    hipMalloc((void**)&d_v2,N*sizeof(int));
    hipMalloc((void**)&d_res,N*sizeof(int));

    for(int i=0;i<N;i++){
        v1[i] = i;
        v2[i] = 2*i;
    }

    hipMemcpy(d_v1,v1,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_v2,v2,N*sizeof(int),hipMemcpyHostToDevice);  

    int numBlocks = (N+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    add_vectors<<<numBlocks,THREADS_PER_BLOCK>>>(d_v1,d_v2,d_res,N);

    hipMemcpy(res,d_res,N*sizeof(int),hipMemcpyDeviceToHost);
    printf("Resultant vector using %d blocks (256 threads per block) : \n",numBlocks);
    for(int i=0;i<N;i++){
        printf("%d ",res[i]);
    }
    printf("\n");

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_res);

    free(v1);
    free(v2);
    free(res);

    return 0;
}
