#include "hip/hip_runtime.h"

#include<stdio.h>

#define THREADS_PER_BLOCK 256

__global__ void calc_sine(float *angles,float *sineValues,int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<N) {
        sineValues[i] = sin(angles[i]);
    }
}

int main(void)
{
    int N;
    printf("Enter the value of N : ");
    scanf("%d",&N);

    float *angles = (float*)malloc(N*sizeof(float));    
    float *sineValues = (float*)malloc(N*sizeof(float));

    float *d_angles,*d_sineValues;
    hipMalloc((void**)&d_angles,N*sizeof(float));
    hipMalloc((void**)&d_sineValues,N*sizeof(float));

    for(int i=0;i<N;i++){
       angles[i] = i;
    }

    hipMemcpy(d_angles,angles,N*sizeof(float),hipMemcpyHostToDevice);

    int numBlocks = (N+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    calc_sine<<<numBlocks,THREADS_PER_BLOCK>>>(d_angles,d_sineValues,N);

    hipMemcpy(sineValues,d_sineValues,N*sizeof(float),hipMemcpyDeviceToHost);
    printf("Resultant sine values using %d blocks (256 threads per block) : \n",numBlocks);
    for(int i=0;i<N;i++){
        printf("%f ",sineValues[i]);
    }
    printf("\n");

    hipFree(d_angles);
    hipFree(d_sineValues);

    free(angles);
    free(sineValues);

    return 0;
}
