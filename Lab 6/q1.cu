#include "hip/hip_runtime.h"

#include<stdio.h>

#define THREADS_PER_BLOCK 256

__global__ void convolution_1D(int width,int mask_width,int *arr,int *mask_arr,int* conv_arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<width){
        int Pvalue = 0;
        int start_point = i-(mask_width/2);
        for(int j=0;j<mask_width;j++){
            if(start_point+j >=0 && start_point+j < width) {
                Pvalue += (arr[start_point+j]*mask_arr[j]);
            }
        }
        conv_arr[i] = Pvalue;
    }
}

int main(void)
{
    int width,mask_width;

    printf("Enter width of the array : ");
    scanf("%d",&width);
    
    printf("Enter mask width : ");
    scanf("%d",&mask_width);
    
    int *arr = (int*)malloc(width*sizeof(int));
    int *mask_arr = (int*)malloc(mask_width*sizeof(int));
    int *conv_arr = (int*)malloc(width*sizeof(int));  
    
    printf("Enter the one-dimensional array : ");
    for(int i=0;i<width;i++){
        scanf("%d",&arr[i]);
    }

    printf("Enter the mask array : ");
    for(int i=0;i<mask_width;i++){
        scanf("%d",&mask_arr[i]);
    }

    int *d_arr,*d_mask_arr,*d_conv_arr;
    hipMalloc((void**)&d_arr,width*sizeof(int));
    hipMalloc((void**)&d_conv_arr,width*sizeof(int));
    hipMalloc((void**)&d_mask_arr,mask_width*sizeof(int));

    hipMemcpy(d_arr,arr,width*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_mask_arr,mask_arr,mask_width*sizeof(int),hipMemcpyHostToDevice);

    int numBlocks = (width+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    convolution_1D<<<numBlocks,THREADS_PER_BLOCK>>>(width,mask_width,d_arr,d_mask_arr,d_conv_arr);

    hipMemcpy(conv_arr,d_conv_arr,width*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant array after convolution : \n");
    for(int i=0;i<width;i++){
        printf("%d ",conv_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_conv_arr);
    hipFree(d_mask_arr);

    free(arr);
    free(conv_arr);
    free(mask_arr);

    return 0;
}
