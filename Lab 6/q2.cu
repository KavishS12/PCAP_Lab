#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void selectionSort(int *arr, int n) {
    int idx = blockIdx.x;
    if (idx < n) {
        for (int i=idx;i<n-1;i++) {
            int minIndex = i;
            for (int j=i+1;j<n;j++) {
                if(arr[j]<arr[minIndex]) {
                    minIndex = j;
                }
            }
            if (minIndex != i) {
                int temp = arr[i];
                arr[i] = arr[minIndex];
                arr[minIndex] = temp;
            }
        }
    }
}

int main() {
    int N;
    printf("Enter size of the array : ");
    scanf("%d",&N);

    int *arr = (int*)malloc(N*sizeof(int));

    printf("Enter the array : ");
    for(int i=0;i<N;i++){
        scanf("%d",&arr[i]);
    }

    int *d_arr;
    hipMalloc((void**)&d_arr,N*sizeof(int));
    hipMemcpy(d_arr,arr,N*sizeof(int),hipMemcpyHostToDevice);

    selectionSort<<<N,1>>>(d_arr,N);

    hipMemcpy(arr,d_arr,N*sizeof(int),hipMemcpyDeviceToHost);

    printf("Sorted Array:\n");
    for (int i=0;i<N;i++) {
        printf("%d ",arr[i]);
    }
    printf("\n");

    hipFree(d_arr);

    return 0;
}
