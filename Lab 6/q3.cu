#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define THREADS_PER_BLOCK 16

__global__ void oddPhase(int *d_array, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2 == 1 && idx < n - 1) {
        if (d_array[idx] > d_array[idx + 1]) {
            int temp = d_array[idx];
            d_array[idx] = d_array[idx + 1];
            d_array[idx + 1] = temp;
        }
    }
}

__global__ void evenPhase(int *d_array, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2 == 0 && idx < n - 1) {
        if (d_array[idx] > d_array[idx + 1]) {
            int temp = d_array[idx];
            d_array[idx] = d_array[idx + 1];
            d_array[idx + 1] = temp;
        }
    }
}

int main() {
    int N;
    printf("Enter size of the array : ");
    scanf("%d",&N);

    int *arr = (int*)malloc(N*sizeof(int));

    printf("Enter the array : ");
    for(int i=0;i<N;i++){
        scanf("%d",&arr[i]);
    }

    int *d_arr;
    hipMalloc((void **)&d_arr,N*sizeof(int));
    hipMemcpy(d_arr,arr,N*sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    for (int i=0;i<N;i++) {
        oddPhase<<<numBlocks, THREADS_PER_BLOCK>>>(d_arr,N);
        hipDeviceSynchronize();
        evenPhase<<<numBlocks, THREADS_PER_BLOCK>>>(d_arr,N);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr,N*sizeof(int),hipMemcpyDeviceToHost);

    printf("Sorted Array:\n");
    for(int i=0;i<N;i++){
        printf("%d ",arr[i]);
    }
    printf("\n");

    hipFree(d_arr);

    return 0;
}

