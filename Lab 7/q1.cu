#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void cuda_count_word(char *sentence,char *word,unsigned int *d_count,int slen,int wlen) {
    int i = threadIdx.x;
    if (i < slen) {
        int match = 1;
        for (int j = 0; j < wlen; j++) {
            if (sentence[i+j] != word[j]) {
                match = 0;
                break;
            }
        }
        if (match == 1) {
            atomicAdd(d_count, 1);
        }
    }
}

int main() {
    char sentence[N];
    char word[N];
    char *d_sentence, *d_word;
    unsigned int *count = (unsigned int*)malloc(sizeof(unsigned int)),*d_count;
    *count = 0;

    printf("Enter a sentence: ");
    fgets(sentence, N, stdin);

    printf("Enter the word to search: ");
    scanf("%s", word);

    int slen = strlen(sentence);
    int wlen = strlen(word);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_sentence,(slen+1)*sizeof(char));
    hipMalloc((void**)&d_word, (wlen+1)*sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(d_sentence,sentence,(slen+1)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_word,word,(wlen+1)*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(unsigned int),hipMemcpyHostToDevice);

    cuda_count_word<<<1, slen>>>(d_sentence, d_word, d_count, slen, wlen);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Cuda error: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipMemcpy(count,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);

    printf("Total occurrences of the word '%s': %u\n",word,*count);
    printf("Time taken: %f ms\n", elapsed_time);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);
    free(count);

    return 0;
}
