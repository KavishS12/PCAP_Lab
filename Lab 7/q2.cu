#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void repeatStringKernel(const char* str_S,char* str_RS,int lenS,int output_len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int start = idx*(2*lenS+(idx-1)*(-1))/2;
    for(int i =0; i<lenS-idx; i++){
        str_RS[start+i] = str_S[i];
    }
}

int main() {
    char str_S[N];
    printf("Enter a string : ");
    scanf("%s",str_S);

    int lenS = strlen(str_S);
    int output_len = (lenS * (lenS+1))/2;

    char* str_RS = (char*)malloc((output_len + 1) * sizeof(char));
    str_RS[output_len] = '\0';

    char *d_S, *d_str_RS;
    hipMalloc((void**)&d_S, lenS * sizeof(char));
    hipMalloc((void**)&d_str_RS, (output_len + 1) * sizeof(char));

    hipMemcpy(d_S, str_S, lenS * sizeof(char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (output_len + threadsPerBlock - 1) / threadsPerBlock;
    repeatStringKernel<<<blocksPerGrid, threadsPerBlock>>>(d_S, d_str_RS, lenS, output_len);

    hipMemcpy(str_RS, d_str_RS, (output_len + 1) * sizeof(char), hipMemcpyDeviceToHost);


    printf("Input string S: %s\n", str_S);
    printf("Output string str_RS: %s\n", str_RS);

    hipFree(d_S);
    hipFree(d_str_RS);
    free(str_RS);

    return 0;
}