#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void cuda_count(char *str,unsigned int *d_count){
    int i = threadIdx.x;
    if(str[i]=='a'){
        atomicAdd(d_count,1);
    }
}

int main(){
    char str[N];
    char *d_str;
    unsigned int *count = (unsigned int*)malloc(sizeof(unsigned int)), *d_count;
    *count = 0;

    printf("Enter a string : ");
    scanf("%s",str);

    hipEvent_t start, stop;
    // Create the events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record the start event
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_str,(strlen(str)+1)*sizeof(char));
    hipMalloc((void**)&d_count,sizeof(unsigned int));

    // Initialize d_count to 0 on the device for atomic operation
    hipMemcpy(d_count, count, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_str,str,(strlen(str)+1)*sizeof(char),hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("Cuda error : %s\n",hipGetErrorString(error));
    }   
    
    cuda_count<<<1,strlen(str)>>>(d_str,d_count);

    error = hipGetLastError();
    if(error != hipSuccess){
        printf("Cuda error : %s\n",hipGetErrorString(error));
    }  

    //Record the stop event
    hipEventRecord(stop, 0);
    // Wait for the stop event to complete
    hipEventSynchronize(stop);
    // Calculate the time between start and stop events
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipMemcpy(count,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);

    printf("Total occurences : %u\n",*count);
    printf("Time taken : %f ms\n",elapsed_time);

    hipFree(d_str);
    hipFree(d_count);
    free(count);

    return 0;

}