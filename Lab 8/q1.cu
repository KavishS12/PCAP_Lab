#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void addRowise(int *mat1,int *mat2,int *res,int n){
    int i = threadIdx.x;
    int m = blockDim.x;
    if(i<m){
        for(int j=0;j<n;j++){
            res[i*n+j] = mat1[i*n+j] + mat2[i*n+j];
        }
    }
}

__global__ void addColumnwise(int *mat1,int *mat2,int *res,int m){
    int j = threadIdx.x;
    int n = blockDim.x;
    if(j<n) {
        for(int i=0;i<m;i++){
            res[i*n+j] = mat1[i*n+j] + mat2[i*n+j];
        }
    }
}

__global__ void addElementwise(int *mat1,int *mat2,int *res){
    int i = blockIdx.x;
    int j = threadIdx.x;
    int m = gridDim.x;
    int n = blockDim.x;
    if(i<m && j<n) {
        res[i*n+j] = mat1[i*n+j] + mat2[i*n+j];
    }
}

int main()
{
    int *mat1,*mat2,*res,m,n;
    int *d_mat1,*d_mat2,*d_res;

    printf("Enter the value of m : ");
    scanf("%d",&m);
    printf("Enter the value of n : ");
    scanf("%d",&n);

    int size = m*n*sizeof(int);
    mat1 = (int*)malloc(size);
    mat2 = (int*)malloc(size);
    res = (int*)malloc(size);

    printf("Enter the first matrix : \n");
    for(int i=0;i<m*n;i++){
        scanf("%d",&mat1[i]);
    }

    printf("Enter the second matrix : \n");
    for(int i=0;i<m*n;i++){
        scanf("%d",&mat2[i]);
    }

    hipMalloc((void**)&d_mat1,size);
    hipMalloc((void**)&d_mat2,size);
    hipMalloc((void**)&d_res,size);

    hipMemcpy(d_mat1,mat1,size,hipMemcpyHostToDevice);
    hipMemcpy(d_mat2,mat2,size,hipMemcpyHostToDevice);

    //addRowise<<<1,m>>> (d_mat1,d_mat2,d_res,n);
    //addColumnwise<<<1,n>>>(d_mat1,d_mat2,d_res,m);
    addElementwise<<<m,n>>> (d_mat1,d_mat2,d_res);

    hipMemcpy(res,d_res,size,hipMemcpyDeviceToHost);

    printf("Resultant matrix : \n");
    for(int i=0;i<m;i++){
        for(int j=0;j<n;j++){
            printf("%d ",res[i*n+j]);
        }
        printf("\n");
    }

    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_res);

    free(mat1);
    free(mat2);
    free(res);

    return 0;

}