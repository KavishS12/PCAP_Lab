#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void multiplyRowwise(int *mat1, int *mat2, int *res, int n1, int n2)
{
    int ridA = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (ridA < n1) {
        for (int cidB=0;cidB<n2;cidB++) {
            sum = 0;
            for (int k=0;k<n1;k++) {
                sum += (mat1[ridA*n1+k] * mat2[k*n2+cidB]);
            }
            res[ridA*n2+cidB] = sum;
        }
    }
}

__global__ void multiplyColumnwise(int *mat1, int *mat2, int *res, int m1, int n1)
{
    int cidB = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (cidB < n1) {
        for (int ridA = 0; ridA < m1; ridA++) {
            sum = 0;
            for (int k = 0; k < n1; k++) {
                sum += (mat1[ridA*n1+k] * mat2[k*n1+cidB]);
            }
            res[ridA*n1+cidB] = sum;
        }
    }
}

__global__ void multiplyElementwise(int *mat1, int *mat2, int *res, int n1)
{
    int ridA = blockIdx.y * blockDim.y + threadIdx.y;
    int cidB = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (ridA < n1 && cidB < n1) {
        for (int k = 0; k < n1; k++) {
            sum += (mat1[ridA*n1+k] * mat2[k*n1+cidB]);
        }
        res[ridA*n1+cidB] = sum;
    }
}


int main()
{
    int *mat1,*mat2,*res,m1,n1,m2,n2;
    int *d_mat1,*d_mat2,*d_res;

    printf("Enter the value of m1 : ");
    scanf("%d",&m1);
    printf("Enter the value of n1 : ");
    scanf("%d",&n1);

    printf("Enter the value of m2 : ");
    scanf("%d",&m2);
    printf("Enter the value of n2 : ");
    scanf("%d",&n2);

    if(m2 != n1) {
        printf("Dimensions do not match. Aborting...\n");
        fflush(stdout); 
        abort();
    }

    mat1 = (int*)malloc(m1*n1*sizeof(int));
    mat2 = (int*)malloc(m2*n2*sizeof(int));
    res = (int*)malloc(m1*n2*sizeof(int));

    printf("Enter the first matrix : \n");
    for(int i=0;i<m1*n1;i++){
        scanf("%d",&mat1[i]);
    }

    printf("Enter the second matrix : \n");
    for(int i=0;i<m2*n2;i++){
        scanf("%d",&mat2[i]);
    }

    hipMalloc((void**)&d_mat1,m1*n1*sizeof(int));
    hipMalloc((void**)&d_mat2,m2*n2*sizeof(int));
    hipMalloc((void**)&d_res,m1*n2*sizeof(int));

    hipMemcpy(d_mat1,mat1,m1*n1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_mat2,mat2,m2*n2*sizeof(int),hipMemcpyHostToDevice);

    //multiplyRowwise<<<(m1+255)/256,256>>>(d_mat1, d_mat2, d_res, n1, n2);
    //multiplyColumnwise<<<(n2+255)/256,256>>>(d_mat1, d_mat2, d_res, m1, n1);
    multiplyElementwise<<<dim3((n2+15)/16,(m1+15)/16),dim3(16,16)>>>(d_mat1, d_mat2, d_res, n1);

    hipMemcpy(res,d_res,m1*n2*sizeof(int),hipMemcpyDeviceToHost);

    printf("Resultant matrix : \n");
    for(int i=0;i<m1;i++){
        for(int j=0;j<n2;j++){
            printf("%d ",res[i*n2+j]);
        }
        printf("\n");
    }

    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_res);

    free(mat1);
    free(mat2);
    free(res);

    return 0;

}