#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 3
#define N 4

__global__ void modify_matrix_kernel(float *A, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        float val = A[row * cols + col];
        for (int i = 0; i < row; i++) { 
            val *= A[row * cols + col];
        }
        A[row * cols + col] = val;
    }
}

int main() {
    float A[M][N] = {{1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}};
    float *d_A;

    hipMalloc((void **)&d_A, M * N * sizeof(float));

    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, M);
    dim3 blocksPerGrid(1, 1);
    modify_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, M, N);

    hipMemcpy(A, d_A, M * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Modified matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%.2f ", A[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);

    return 0;
}