#include <stdio.h>
#include <hip/hip_runtime.h>

#define M 4
#define N 4

__global__ void complement_kernel(int *A, int *B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        if (row > 0 && row < rows - 1 && col > 0 && col < cols - 1) {
            B[row * cols + col] = ~A[row * cols + col];
        } else {
            B[row * cols + col] = A[row * cols + col];
        }
    }
}

int main() {
    int A[M][N] = {{1, 2, 3, 4}, {6, 5, 8, 3}, {2, 4, 10, 1}, {9, 1, 2, 5}};
    int B[M][N] = {0};
    int *d_A, *d_B;

    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(4, 4);
    dim3 blocksPerGrid(1, 1);
    complement_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, M, N);

    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Output matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}